#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
#include <math.h>

using namespace std;

__global__ void max(int* input, int n)
{
	const int tid = threadIdx.x;

	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) 
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			if(snd < n)
			{
				if(input[snd] > input[fst])
					input[fst] = input[snd];
			}
		}

		step_size *= 2; 
		if(number_of_threads == 1)
			break;
		number_of_threads = (int)ceil((float)number_of_threads/2.0);      // divide number of threads by 2
		__syncthreads();
	}
}

int main()
{
	int count;
	int result;
	int* d;

	cout<<"\nEnter the number of elements : ";
	cin>>count;
	const int size = count * sizeof(int);

	int *h;
	h = new int[count];

	cout<<"\nEnter the elements : \n";
	for(int i=0;i<count;i++)
		cin>>h[i];
	//h[i] = rand()%1000
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	max <<<1, ceil((float)count/2.0) >>>(d , count);

	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Max is " << result << endl;

	getchar();

	hipFree(d);
	delete[] h;

	return 0;
}

/*
PS D:\MyFiles\Projects\LP1-LabAsg\1-HPC> nvcc ParRedMax.cu -o ParRedMax
ParRedMax.cu
   Creating library ParRedMax.lib and object ParRedMax.exp
PS D:\MyFiles\Projects\LP1-LabAsg\1-HPC> nvprof ./ParRedMax

Enter the number of elements : 4

Enter the elements :
67
3
78
32
==10688== NVPROF is profiling process 10688, command: ./ParRedMax
Max is 78
==10688== Profiling application: ./ParRedMax
==10688== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   60.33%  2.4330us         1  2.4330us  2.4330us  2.4330us  max(int*, int)
                   25.39%  1.0240us         1  1.0240us  1.0240us  1.0240us  [CUDA memcpy HtoD]
                   14.28%     576ns         1     576ns     576ns     576ns  [CUDA memcpy DtoH]
      API calls:   74.44%  154.08ms         1  154.08ms  154.08ms  154.08ms  cudaMalloc
                   25.25%  52.275ms         1  52.275ms  52.275ms  52.275ms  cuDevicePrimaryCtxRelease
                    0.13%  277.30us        97  2.8580us     100ns  178.30us  cuDeviceGetAttribute
                    0.06%  130.30us         1  130.30us  130.30us  130.30us  cudaFree
                    0.04%  82.600us         2  41.300us  24.900us  57.700us  cudaMemcpy
                    0.04%  80.300us         1  80.300us  80.300us  80.300us  cuModuleUnload
                    0.01%  28.900us         1  28.900us  28.900us  28.900us  cudaLaunchKernel
                    0.01%  27.100us         1  27.100us  27.100us  27.100us  cuDeviceTotalMem
                    0.00%  9.5000us         1  9.5000us  9.5000us  9.5000us  cuDeviceGetPCIBusId
                    0.00%  2.3000us         2  1.1500us     200ns  2.1000us  cuDeviceGet
                    0.00%  2.0000us         3     666ns     300ns     900ns  cuDeviceGetCount
                    0.00%     800ns         1     800ns     800ns     800ns  cuDeviceGetName
                    0.00%     400ns         1     400ns     400ns     400ns  cuDeviceGetLuid
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetUuid

*/