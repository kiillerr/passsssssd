#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
#include <math.h>

using namespace std;

__global__ void min(int* input, int n)
{
	const int tid = threadIdx.x; //Index of the thread within the block

	int step_size = 1;
	int number_of_threads = blockDim.x; //Number of threads in thread block

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) 
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			if(snd < n)
			{
				if(input[snd] < input[fst])
					input[fst] = input[snd];
			}
		}

		step_size <<= 1; //1 -> 2, 2 -> 4, 3->6; shift operator
		if(number_of_threads == 1)
			break;
		number_of_threads = (int)ceil((float)number_of_threads/2.0);      // divide number of threads by 2
		__syncthreads();
	}
}

int main()
{
	int count;
	int result;
	int* d;

	cout<<"\nEnter the number of elements : ";
	cin>>count;
	const int size = count * sizeof(int);
	//when sizeof() is used with data types it simply returns the
	// the amount of memory allocated to that data types
	int *h;
	h = new int[count];

	cout<<"\nEnter the elements : \n";
	for(int i=0;i<count;i++)
		cin>>h[i];
	

	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	min <<<1, ceil((float)count/2.0) >>>(d , count);

	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Min is " << result << endl;

	getchar();

	hipFree(d);
	delete[] h;

	return 0;
}

/* 
PS D:\MyFiles\Projects\LP1-LabAsg\1-HPC> nvcc ParRedMin.cu -o ParRedMin
ParRedMin.cu
   Creating library ParRedMin.lib and object ParRedMin.exp
PS D:\MyFiles\Projects\LP1-LabAsg\1-HPC> nvprof ./ParRedMin

Enter the number of elements : 4

Enter the elements :
1
2
3
67
==1876== NVPROF is profiling process 1876, command: ./ParRedMin
Min is 1
==1876== Profiling application: ./ParRedMin
==1876== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   59.68%  2.3680us         1  2.3680us  2.3680us  2.3680us  min(int*, int)
                   25.81%  1.0240us         1  1.0240us  1.0240us  1.0240us  [CUDA memcpy HtoD]
                   14.52%     576ns         1     576ns     576ns     576ns  [CUDA memcpy DtoH]
      API calls:   79.80%  172.77ms         1  172.77ms  172.77ms  172.77ms  cudaMalloc
                   19.78%  42.826ms         1  42.826ms  42.826ms  42.826ms  cuDevicePrimaryCtxRelease
                    0.13%  271.80us        97  2.8020us     100ns  172.50us  cuDeviceGetAttribute
                    0.11%  245.30us         1  245.30us  245.30us  245.30us  cudaLaunchKernel
                    0.07%  144.50us         1  144.50us  144.50us  144.50us  cudaFree
                    0.05%  106.90us         2  53.450us  25.500us  81.400us  cudaMemcpy
                    0.04%  89.600us         1  89.600us  89.600us  89.600us  cuModuleUnload
                    0.01%  21.400us         1  21.400us  21.400us  21.400us  cuDeviceTotalMem
                    0.00%  9.7000us         1  9.7000us  9.7000us  9.7000us  cuDeviceGetPCIBusId
                    0.00%  1.9000us         3     633ns     200ns     900ns  cuDeviceGetCount
                    0.00%  1.7000us         2     850ns     300ns  1.4000us  cuDeviceGet
                    0.00%     900ns         1     900ns     900ns     900ns  cuDeviceGetName
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetUuid
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetLuid

*/