#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"


#include <iostream>
#include <numeric>
#include <math.h>

using namespace std;

#define BLOCK_SIZE 4;

__global__ void mean(float* input, int n)
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) 
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			if(snd < n)
			{
				input[fst] += input[snd];// a = a+b
			}				
		}

		step_size <<= 1; 
		if(number_of_threads == 1)
			break;
		number_of_threads = (int)ceil((float)number_of_threads/2.0);      // divide number of threads by 2
		__syncthreads();
	}

	__syncthreads();
	input[0] /= n;
}

int main()
{
	int count=0;
	float result;
	float *d;

	cout<<"\nEnter the number of elements : ";
	cin>>count;
	const int size = count * sizeof(float);

	float *h;
	h = new float[count];

	cout<<"\nEnter the elements : \n";
	for(int i=0;i<count;i++)
		cin>>h[i];
	//h[i] = rand()%1000;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	//cout<<ceil((float)count/2.0);
	mean <<<1, ceil((float)count/2.0) >>>(d,count);

	
	hipMemcpy(&result, d, sizeof(float), hipMemcpyDeviceToHost);

	cout << "Mean is " << result << endl;

	getchar();

	hipFree(d);
	delete[] h;

	return 0;
}

/*
PS D:\MyFiles\Projects\LP1-LabAsg\2-HPC> nvcc ParRedMean.cu -o ParRedMean
ParRedMean.cu
   Creating library ParRedMean.lib and object ParRedMean.exp
PS D:\MyFiles\Projects\LP1-LabAsg\2-HPC> nvprof ./ParRedMean

Enter the number of elements : 4

Enter the elements :
2
3
6
1
==26012== NVPROF is profiling process 26012, command: ./ParRedMean
Mean is 3
==26012== Profiling application: ./ParRedMean
==26012== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   63.04%  2.7840us         1  2.7840us  2.7840us  2.7840us  mean(float*, int)
                   23.91%  1.0560us         1  1.0560us  1.0560us  1.0560us  [CUDA memcpy HtoD]
                   13.04%     576ns         1     576ns     576ns     576ns  [CUDA memcpy DtoH]
      API calls:   78.89%  167.23ms         1  167.23ms  167.23ms  167.23ms  cudaMalloc
                   20.69%  43.855ms         1  43.855ms  43.855ms  43.855ms  cuDevicePrimaryCtxRelease
                    0.14%  293.70us        97  3.0270us     100ns  163.00us  cuDeviceGetAttribute
                    0.11%  226.30us         1  226.30us  226.30us  226.30us  cudaFree
                    0.08%  167.00us         1  167.00us  167.00us  167.00us  cuModuleUnload
                    0.05%  116.30us         2  58.150us  22.700us  93.600us  cudaMemcpy
                    0.03%  61.100us         1  61.100us  61.100us  61.100us  cuDeviceTotalMem
                    0.01%  28.300us         1  28.300us  28.300us  28.300us  cudaLaunchKernel
                    0.00%  10.000us         1  10.000us  10.000us  10.000us  cuDeviceGetPCIBusId
                    0.00%  1.5000us         3     500ns     300ns     900ns  cuDeviceGetCount
                    0.00%  1.2000us         2     600ns     100ns  1.1000us  cuDeviceGet
                    0.00%     700ns         1     700ns     700ns     700ns  cuDeviceGetName
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetUuid
                    0.00%     300ns         1     300ns     300ns     300ns  cuDeviceGetLuid
*/