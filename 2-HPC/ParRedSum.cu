#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
#include <math.h>

using namespace std;

#define BLOCK_SIZE 4;

__global__ void sum(int* input, int n)                                      // global call to cuda function (host to device)
{
	const int tid = threadIdx.x;                                            // get thread ID
	int step_size = 1;
	int number_of_threads = blockDim.x;                                     // initiate step size and number of threads 

	while (number_of_threads > 0)                                          
	{
		if (tid < number_of_threads) 
		{
			const int fst = tid * step_size * 2;                           
			const int snd = fst + step_size;                                // calculate indices of first and second element to be added
			if(snd < n)
			{
				input[fst] += input[snd];                                   // add elements
			}
		}

		step_size <<= 1;                                                   // multiply step size by 2                                  
		if(number_of_threads == 1)
			break;
		number_of_threads = (int)ceil((float)number_of_threads/2.0);      // divide number of threads by 2
		__syncthreads();
	}
}

int main()
{
	int count=0;
	int result;
	int *d;

	cout<<"\nEnter the number of elements : ";
	cin>>count;
	const int size = count * sizeof(int);

	int *h;
	h = new int[count];

	cout<<"\nEnter the elements : \n";
	for(int i=0;i<count;i++)
		cin>>h[i];
	
	hipMalloc(&d, size);                                                  // allocate device variable memory
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);                        // copy array from host to device

	//cout<<ceil((float)count/2.0);
	sum <<<1, ceil((float)count/2.0) >>>(d,count);                         // function call  func_name<<<no_of_blocks,no_of_threads>>>(args)

	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);           // copy result back from device to host

	cout << "Sum is " << result << endl;

	getchar();

	hipFree(d);                                                           // free device memory
	delete[] h;

	return 0;
}

/*
PS D:\MyFiles\Projects\LP1-LabAsg\2-HPC> nvcc ParRedSum.cu -o ParRedSum
ParRedSum.cu
   Creating library ParRedSum.lib and object ParRedSum.exp
PS D:\MyFiles\Projects\LP1-LabAsg\2-HPC> nvprof ./ParRedSum

Enter the number of elements : 4

Enter the elements :
2
49
12
54
==4900== NVPROF is profiling process 4900, command: ./ParRedSum
Sum is 117
==4900== Profiling application: ./ParRedSum
==4900== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   60.16%  2.4640us         1  2.4640us  2.4640us  2.4640us  sum(int*, int)
                   25.00%  1.0240us         1  1.0240us  1.0240us  1.0240us  [CUDA memcpy HtoD]
                   14.84%     608ns         1     608ns     608ns     608ns  [CUDA memcpy DtoH]
      API calls:   82.75%  203.24ms         1  203.24ms  203.24ms  203.24ms  cudaMalloc
                   16.83%  41.338ms         1  41.338ms  41.338ms  41.338ms  cuDevicePrimaryCtxRelease
                    0.16%  392.40us        97  4.0450us     100ns  220.50us  cuDeviceGetAttribute
                    0.10%  243.20us         1  243.20us  243.20us  243.20us  cudaFree
                    0.07%  170.80us         2  85.400us  62.900us  107.90us  cudaMemcpy
                    0.06%  151.20us         1  151.20us  151.20us  151.20us  cuModuleUnload
                    0.01%  29.200us         1  29.200us  29.200us  29.200us  cudaLaunchKernel
                    0.01%  18.800us         1  18.800us  18.800us  18.800us  cuDeviceTotalMem
                    0.00%  9.8000us         1  9.8000us  9.8000us  9.8000us  cuDeviceGetPCIBusId
                    0.00%  1.3000us         3     433ns     200ns     800ns  cuDeviceGetCount
                    0.00%     900ns         1     900ns     900ns     900ns  cuDeviceGetName
                    0.00%     700ns         2     350ns     100ns     600ns  cuDeviceGet
                    0.00%     400ns         1     400ns     400ns     400ns  cuDeviceGetLuid
                    0.00%     200ns         1     200ns     200ns     200ns  cuDeviceGetUuid
*/